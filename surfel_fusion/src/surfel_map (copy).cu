#include "hip/hip_runtime.h"
#include <surfel_map.h>
// #include <cuda_functions.cuh>
#include <timer.h>
#include <algorithm>
#include <pcl/io/pcd_io.h>

SurfelMap::SurfelMap(ros::NodeHandle &_nh):
nh(_nh),
// fuse_param_gpuptr(NULL),
inactive_pointcloud(new PointCloud)
{
    // get the parameters
    bool get_all = true;
    get_all &= nh.getParam("cam_width", cam_width);
    get_all &= nh.getParam("cam_height", cam_height);
    get_all &= nh.getParam("cam_fx", cam_fx);
    get_all &= nh.getParam("cam_cx", cam_cx);
    get_all &= nh.getParam("cam_fy", cam_fy);
    get_all &= nh.getParam("cam_cy", cam_cy);
    
    // get extrinsic params
    get_all &= nh.getParam("ric00", Ric00);
    get_all &= nh.getParam("ric01", Ric01);
    get_all &= nh.getParam("ric02", Ric02);
    get_all &= nh.getParam("ric10", Ric10);
    get_all &= nh.getParam("ric11", Ric11);
    get_all &= nh.getParam("ric12", Ric12);
    get_all &= nh.getParam("ric20", Ric20);
    get_all &= nh.getParam("ric21", Ric21);
    get_all &= nh.getParam("ric22", Ric22);

    get_all &= nh.getParam("tic0", Tic0);
    get_all &= nh.getParam("tic1", Tic1);
    get_all &= nh.getParam("tic2", Tic2);


/*    imu_cam_rot << Ric00, Ric01, Ric02,
                   Ric10, Ric11, Ric12,
                   Ric20, Ric21, Ric22;

    imu_cam_tra << Tic0, Tic1, Tic2;*/

    // imu_cam_rot = imu_cam_rot.transpose();
    // imu_cam_tra = - imu_cam_rot * imu_cam_tra;

    camera_matrix = Eigen::Matrix3d::Zero();
    camera_matrix(0, 0) = cam_fx;
    camera_matrix(0, 2) = cam_cx;
    camera_matrix(1, 1) = cam_fy;
    camera_matrix(1, 2) = cam_cy;
    camera_matrix(2, 2) = 1.0;

    get_all &= nh.getParam("fuse_far_distence", far_dist);
    get_all &= nh.getParam("fuse_near_distence", near_dist);
    get_all &= nh.getParam("drift_free_poses", drift_free_poses);

    if(!get_all)
        printf("ERROR! Do not have enough parameters!");
    else
    {
        printf("Have the following settings: \n");
        printf("camera matrix: \n");
        cout << camera_matrix << endl;
        printf("fuse the distence between %4f m and %4f m.\n", near_dist, far_dist);
    }

    // fuse_param.fx = cam_fx;
    // fuse_param.fy = cam_fy;
    // fuse_param.cx = cam_cx;
    // fuse_param.cy = cam_cy;
    // fuse_param.width = cam_width;
    // fuse_param.height = cam_height;
    // fuse_param.far_dist = far_dist;
    // fuse_param.near_dist = near_dist;
    // // local_loop_warp = Eigen::Matrix4d::Identity();
    // hipMalloc(&fuse_param_gpuptr, sizeof(FuseParameters));
    // hipMemcpy(fuse_param_gpuptr, &fuse_param, sizeof(FuseParameters), hipMemcpyHostToDevice);

    fusion_functions.initialize(cam_width, cam_height, cam_fx, cam_fy, cam_cx, cam_cy, far_dist, near_dist);

    // ros publisher
    pointcloud_publish = nh.advertise<PointCloud>("pointcloud", 10);
    raw_pointcloud_publish = nh.advertise<PointCloud>("raw_pointcloud", 10);
    loop_path_publish = nh.advertise<nav_msgs::Path>("fusion_loop_path", 10);
    driftfree_path_publish = nh.advertise<visualization_msgs::Marker>("driftfree_loop_path", 10);
    loop_marker_publish = nh.advertise<visualization_msgs::Marker>("loop_marker", 10);

    // render_tool initialize
    render_tool.initialize_rendertool(cam_width, cam_height, cam_fx, cam_fy, cam_cx, cam_cy);

    //
    is_first_path = true;
    extrinsic_matrix_initialized = false;
}

SurfelMap::~SurfelMap()
{
    // if (fuse_param_gpuptr)
    //     hipFree(fuse_param_gpuptr);
}

void SurfelMap::save_map(const std_msgs::StringConstPtr &save_map_input)
{
    string save_name = save_map_input->data;
    printf("save mesh modelt to %s.\n", save_name.c_str());
    save_mesh(save_name);
    printf("save done!\n");
}

void SurfelMap::image_input(const sensor_msgs::ImageConstPtr &image_input)
{
    // printf("receive image!\n");
    cv_bridge::CvImagePtr image_ptr = cv_bridge::toCvCopy(image_input, sensor_msgs::image_encodings::MONO8);
    cv::Mat image = image_ptr->image;
    ros::Time stamp = image_ptr->header.stamp;
    image_buffer.push_back(std::make_pair(stamp, image));
    synchronize_msgs();
}

void SurfelMap::depth_input(const sensor_msgs::ImageConstPtr &depth_input)
{
    // printf("receive depth!\n");
    cv_bridge::CvImagePtr image_ptr;
    image_ptr = cv_bridge::toCvCopy(depth_input, depth_input->encoding);
    constexpr double kDepthScalingFactor = 0.001;
    if(depth_input->encoding == sensor_msgs::image_encodings::TYPE_16UC1)
        (image_ptr->image).convertTo(image_ptr->image, CV_32FC1, kDepthScalingFactor);
    // image_ptr = cv_bridge::toCvCopy(depth_input, sensor_msgs::image_encodings::TYPE_32FC1);
    cv::Mat image = image_ptr->image;
    ros::Time stamp = image_ptr->header.stamp;
    depth_buffer.push_back(std::make_pair(stamp, image));
    synchronize_msgs();
}

void SurfelMap::synchronize_msgs()
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> total_time;
    start_time = std::chrono::system_clock::now();

    if(pose_reference_buffer.size() == 0)
        return;
    
    for(int scan_pose = 0; scan_pose < pose_reference_buffer.size(); scan_pose++)
    {
        ros::Time fuse_stamp = pose_reference_buffer[scan_pose].first;
        double pose_reference_time = fuse_stamp.toSec();
        int image_num = -1;
        int depth_num = -1;
        for(int image_i = 0; image_i < image_buffer.size(); image_i++)
        {
            double image_time = image_buffer[image_i].first.toSec();
            if(fabs(image_time - pose_reference_time) < 0.01)
            {
                image_num = image_i;
            }
        }
        for(int depth_i = 0; depth_i < depth_buffer.size(); depth_i++)
        {
            double depth_time = depth_buffer[depth_i].first.toSec();
            if(fabs(depth_time - pose_reference_time) < 0.01)
            {
                depth_num = depth_i;
            }
        }

        if( image_num < 0 || depth_num < 0)
            continue;

        int relative_index = pose_reference_buffer[scan_pose].second;
        geometry_msgs::Pose fuse_pose = poses_database[relative_index].cam_pose;
        Eigen::Matrix4d fuse_pose_eigen;
        pose_ros2eigen(fuse_pose, fuse_pose_eigen);

        move_add_surfels(relative_index);

        // fuse the current image/depth
        printf("fuse map begins!\n");
        cv::Mat image, depth;
        image = image_buffer.front().second;
        depth = depth_buffer.front().second;
        fuse_map(image, depth, fuse_pose_eigen.cast<float>(), relative_index);
        printf("fuse map done!\n");

        move_all_surfels();

        for(int delete_pose = 0; delete_pose <= scan_pose; delete_pose ++)
            pose_reference_buffer.pop_front();
        for(int delete_image = 0; delete_image <= image_num; delete_image++)
            image_buffer.pop_front();
        for(int delete_depth = 0; delete_depth <= depth_num; delete_depth++)
            depth_buffer.pop_front();

        // {
        //     // debug print the pose value
        //     printf("print the pose value\n");
        //     for(int i = 0; i < poses_database.size(); i++)
        //     {
        //         printf("\nthe pose %d, pose: (%f, %f, %f, %f) position (%f, %f, %f)", i,
        //             poses_database[i].cam_pose.orientation.x,
        //             poses_database[i].cam_pose.orientation.y,
        //             poses_database[i].cam_pose.orientation.z,
        //             poses_database[i].cam_pose.orientation.w,
        //             poses_database[i].cam_pose.position.x,
        //             poses_database[i].cam_pose.position.y,
        //             poses_database[i].cam_pose.position.z
        //         );
        //     }
        //     printf("\n");
        // }
        
        end_time = std::chrono::system_clock::now();
        total_time = end_time - start_time;
        printf("fuse surfels cost %f ms.\n", total_time.count()*1000.0);
        start_time = std::chrono::system_clock::now();    

        // publish results
        publish_raw_pointcloud(depth, image, fuse_pose);
        // publish_neighbor_pointcloud(fuse_stamp, relative_index);
        publish_pose_graph(fuse_stamp, relative_index);
        // render_depth(fuse_pose_ros);
        // if(poses_database.size()%2==0)
        // {
        // publish_all_pointcloud(fuse_stamp);
        // }
        end_time = std::chrono::system_clock::now();
        total_time = end_time - start_time;
        // printf("publish results cost %f ms.\n", total_time.count()*1000.0);
        // calculate_memory_usage();

        // break;
    }
}

void SurfelMap::extrinsic_input(const nav_msgs::OdometryConstPtr &ex_input)
{
    geometry_msgs::Pose ex_pose = ex_input->pose.pose;
    pose_ros2eigen(ex_pose, extrinsic_matrix);
    // std::cout << "receive extrinsic pose" << std::endl <<  extrinsic_matrix << std::endl;
    extrinsic_matrix_initialized = true;
}


void SurfelMap::path_input(const nav_msgs::PathConstPtr &loop_path_input)
{
    if(is_first_path || (!extrinsic_matrix_initialized))
    {
        is_first_path = false;
        pre_path_delete_time = loop_path_input->poses.back().header.stamp.toSec();
        return;
    }

    printf("\nbegin new frame process!!!\n");

    // Eigen::Matrix4d imu2cam = Eigen::Matrix4d::Identity();
    // imu2cam(0,0) = Ric00;
    // imu2cam(0,1) = Ric01;
    // imu2cam(0,2) = Ric02;
    // imu2cam(1,0) = Ric10;
    // imu2cam(1,1) = Ric11;
    // imu2cam(1,2) = Ric12;
    // imu2cam(2,0) = Ric20;
    // imu2cam(2,1) = Ric21;
    // imu2cam(2,2) = Ric22;
    // imu2cam(0,3) = Tic0;
    // imu2cam(1,3) = Tic1;
    // imu2cam(2,3) = Tic2;
    //std::cout << "imu2cam" << std::endl << imu2cam << std::endl;

    nav_msgs::Path camera_path;
    for(int i = 0; i < loop_path_input->poses.size(); i++)
    {
        geometry_msgs::PoseStamped imu_posestamped = loop_path_input->poses[i];
        if(imu_posestamped.header.stamp.toSec() < pre_path_delete_time)
            continue;
        geometry_msgs::PoseStamped cam_posestamped = imu_posestamped;
        Eigen::Matrix4d imu_t, cam_t;
        pose_ros2eigen(imu_posestamped.pose, imu_t);
        cam_t = imu_t * extrinsic_matrix;
        pose_eigen2ros(cam_t, cam_posestamped.pose);
        camera_path.poses.push_back(cam_posestamped);
    }

    bool have_new_pose = false;
    geometry_msgs::Pose input_pose;
    // //geometry_msgs::Pose camera_pose;
    // Eigen::Matrix3d R_wi, R_wc; 
    // Eigen::Vector3d T_wi, T_wc;
    // Eigen::Quaterniond Q_wi, Q_wc; 

    if(camera_path.poses.size() > poses_database.size())
    {
        input_pose = camera_path.poses.back().pose;
        // T_wi << input_pose.position.x, input_pose.position.y, input_pose.position.z;
        // Q_wi.w() = input_pose.orientation.w;
        // Q_wi.x() = input_pose.orientation.x;
        // Q_wi.y() = input_pose.orientation.y;
        // Q_wi.z() = input_pose.orientation.z;
        // R_wi = Q_wi;

        // T_wc = T_wi + R_wi * imu_cam_tra;
        // std::cout << " imu_cam_tra : " << std::endl << imu_cam_tra << std::endl;
        // R_wc = R_wi * imu_cam_rot;
        // Q_wc = R_wc;

        // input_pose.position.x = T_wc.x();
        // input_pose.position.y = T_wc.y();
        // input_pose.position.y = T_wc.z();

        // input_pose.orientation.w = Q_wc.w(); 
        // input_pose.orientation.x = Q_wc.x(); 
        // input_pose.orientation.y = Q_wc.y(); 
        // input_pose.orientation.z = Q_wc.z(); 

        have_new_pose = true;
    }
    
    // first update the poses
    bool loop_changed = false;
    for(int i = 0; i < poses_database.size() && i < camera_path.poses.size(); i++)
    {   
        // input_pose = camera_path.poses[i].pose;

        // T_wi << input_pose2.position.x, input_pose2.position.y, input_pose2.position.z;
        // Q_wi.w() = input_pose2.orientation.w;
        // Q_wi.x() = input_pose2.orientation.x;
        // Q_wi.y() = input_pose2.orientation.y;
        // Q_wi.z() = input_pose2.orientation.z;
        // R_wi = Q_wi;

        // T_wc = T_wi + R_wi * imu_cam_tra;
        // R_wc = R_wi * imu_cam_rot;
        // Q_wc = R_wc;

        // input_pose2.position.x = T_wc.x();
        // input_pose2.position.y = T_wc.y();
        // input_pose2.position.y = T_wc.z();

        // input_pose2.orientation.w = Q_wc.w(); 
        // input_pose2.orientation.x = Q_wc.x(); 
        // input_pose2.orientation.y = Q_wc.y(); 
        // input_pose2.orientation.z = Q_wc.z(); 
        
        poses_database[i].loop_pose = camera_path.poses[i].pose;

        if( poses_database[i].loop_pose.position.x != poses_database[i].cam_pose.position.x
            || poses_database[i].loop_pose.position.y != poses_database[i].cam_pose.position.y
            || poses_database[i].loop_pose.position.z != poses_database[i].cam_pose.position.z)
        {
            loop_changed = true;
        }
    }

    // if(poses_database.size() > camera_path.poses.size())
    // {
    //     int last_update_index = camera_path.poses.size() - 1;
    //     int start_index = camera_path.poses.size();
    //     Eigen::Matrix4d warp_pose, pre_pose, after_pose;
    //     pose_ros2eigen(poses_database[last_update_index].cam_pose, pre_pose);
    //     pose_ros2eigen(poses_database[last_update_index].loop_pose, after_pose);
    //     warp_pose = after_pose * pre_pose.inverse();
    //     for(start_index; start_index < poses_database.size(); start_index++)
    //     {
    //         Eigen::Matrix4d this_pose_pre, this_pose_after;
    //         pose_ros2eigen(poses_database[start_index].cam_pose, this_pose_pre);
    //         this_pose_after = warp_pose * this_pose_pre;
    //         geometry_msgs::Pose after_pose_ros;
    //         pose_eigen2ros(this_pose_after, after_pose_ros);
    //         poses_database[start_index].loop_pose = after_pose_ros;
    //     }
    // }

    printf("warp the surfels according to the loop!\n");
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    start_time = std::chrono::system_clock::now();
    if(loop_changed)
    {
        warp_surfels();
    }
    end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> used_time = end_time - start_time;
    double all_time = used_time.count() * 1000.0;
    printf("warp end! cost %f ms.\n", all_time);

    // // if the current pose is new keyframe
    // bool is_new_keyframe;
    // if(this_pose_input->pose.covariance[0] > 0)
    //     is_new_keyframe = true;
    // else
    //     is_new_keyframe = false;
    // // the corner case that the first frame of the system
    // if(poses_database.size() == 0)
    //     is_new_keyframe = true;
    if(have_new_pose)
    {
        // add new pose
        PoseElement this_pose_element;
        int this_pose_index = poses_database.size();
        this_pose_element.cam_pose = input_pose;
        this_pose_element.loop_pose = input_pose;
        this_pose_element.cam_stamp = camera_path.poses.back().header.stamp;
        if(poses_database.size() > 0)
        {
            int relative_index = poses_database.size() - 1;
            this_pose_element.linked_pose_index.push_back(relative_index);
            poses_database[relative_index].linked_pose_index.push_back(this_pose_index);
        }
        poses_database.push_back(this_pose_element);
        local_surfels_indexs.insert(this_pose_index);
        // printf("add %d keyframe, with pose (%f, %f, %f, %f) and position (%f, %f, %f)!\n",
        //     poses_database.size() - 1,
        //     poses_database.back().cam_pose.orientation.x,
        //     poses_database.back().cam_pose.orientation.y,
        //     poses_database.back().cam_pose.orientation.z,
        //     poses_database.back().cam_pose.orientation.w,
        //     poses_database.back().cam_pose.position.x,
        //     poses_database.back().cam_pose.position.y,
        //     poses_database.back().cam_pose.position.z
        // );

        pose_reference_buffer.push_back(std::make_pair(camera_path.poses.back().header.stamp, this_pose_index));
        synchronize_msgs();
    }

    // push the msg into the buffer for fusion

}

// void SurfelMap::orb_results_input(
//     const sensor_msgs::PointCloudConstPtr &loop_stamp_input,
//     const nav_msgs::PathConstPtr &loop_path_input,
//     const nav_msgs::OdometryConstPtr &this_pose_input)
// {
//     // printf("receive orb message!\n");
//     printf("\nbegin new frame process!!!\n");
//     geometry_msgs::Pose input_pose = this_pose_input->pose.pose;

//     // transform the kitti pose
//     static Eigen::Matrix4d transform_kitti;
//     {
//         Eigen::Matrix4d received_psoe;
//         pose_ros2eigen(input_pose, received_psoe);        
//         if(poses_database.size() == 0)
//         {
//             Eigen::Matrix4d idea_pose;
//             idea_pose = Eigen::Matrix4d::Zero();
//             idea_pose(0,0) = 1.0;
//             idea_pose(1,2) = 1.0;
//             idea_pose(2,1) = -1.0;
//             idea_pose(3,3) = 1.0;
//             transform_kitti = idea_pose * received_psoe.inverse();
//         }
//         Eigen::Matrix4d transformed_pose;
//         transformed_pose = transform_kitti * received_psoe;
//         pose_eigen2ros(transformed_pose, input_pose);
//     }
//     // transform end
    
//     // first update the poses
//     bool loop_changed = false;
//     for(int i = 0; i < poses_database.size() && i < loop_path_input->poses.size(); i++)
//     {
//         poses_database[i].loop_pose = loop_path_input->poses[i].pose;
//         {
//             // transform the kitti pose
//             Eigen::Matrix4d received_pose, transformed_pose;
//             pose_ros2eigen(poses_database[i].loop_pose, received_pose);
//             transformed_pose = transform_kitti *  received_pose;
//             pose_eigen2ros(transformed_pose, poses_database[i].loop_pose);
//         }
//         if( poses_database[i].loop_pose.position.x != poses_database[i].cam_pose.position.x
//             || poses_database[i].loop_pose.position.y != poses_database[i].cam_pose.position.y
//             || poses_database[i].loop_pose.position.z != poses_database[i].cam_pose.position.z)
//         {
//             loop_changed = true;
//         }
//     }

//     if(poses_database.size() > loop_path_input->poses.size())
//     {
//         int last_update_index = loop_path_input->poses.size() - 1;
//         int start_index = loop_path_input->poses.size();
//         Eigen::Matrix4d warp_pose, pre_pose, after_pose;
//         pose_ros2eigen(poses_database[last_update_index].cam_pose, pre_pose);
//         pose_ros2eigen(poses_database[last_update_index].loop_pose, after_pose);
//         warp_pose = after_pose * pre_pose.inverse();
//         for(start_index; start_index < poses_database.size(); start_index++)
//         {
//             Eigen::Matrix4d this_pose_pre, this_pose_after;
//             pose_ros2eigen(poses_database[start_index].cam_pose, this_pose_pre);
//             this_pose_after = warp_pose * this_pose_pre;
//             geometry_msgs::Pose after_pose_ros;
//             pose_eigen2ros(this_pose_after, after_pose_ros);
//             poses_database[start_index].loop_pose = after_pose_ros;
//         }
//     }

//     // if(!all_time_right)
//     //     printf("receive time error!!!!!!!!!!!!!!!!!!!!!\n");
//     // if(poses_database.size() > loop_path_input->poses.size())
//     //     printf("current, we have %d poses and received %d poses.\n ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n",
//     //     poses_database.size(), loop_path_input->poses.size());

//     // warp the surfels

//     printf("warp the surfels according to the loop!\n");
//     std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
//     start_time = std::chrono::system_clock::now();
//     if(loop_changed)
//     {
//         warp_surfels();
//     }
//     end_time = std::chrono::system_clock::now();
//     std::chrono::duration<double> used_time = end_time - start_time;
//     double all_time = used_time.count() * 1000.0;
//     printf("warp end! cost %f ms.\n", all_time);
    

//     // add loop information
//     int loop_num = loop_stamp_input->channels[0].values.size() / 2;
//     for(int i = 0; i < loop_num; i++)
//     {
//         int loop_first = loop_stamp_input->channels[0].values[i*2];
//         int loop_second = loop_stamp_input->channels[0].values[i*2+1];
//         if(loop_first < poses_database.size() && loop_second < poses_database.size())
//         {
//             if(std::find(
//                 poses_database[loop_first].linked_pose_index.begin(),
//                 poses_database[loop_first].linked_pose_index.end(),
//                 loop_second) == poses_database[loop_first].linked_pose_index.end())
//             {
//                 if(std::find(poses_database[loop_first].linked_pose_index.begin(),
//                     poses_database[loop_first].linked_pose_index.end(),
//                     loop_second) == poses_database[loop_first].linked_pose_index.end())
//                     poses_database[loop_first].linked_pose_index.push_back(loop_second);
//                 if(std::find(poses_database[loop_second].linked_pose_index.begin(),
//                     poses_database[loop_second].linked_pose_index.end(),
//                     loop_first) == poses_database[loop_second].linked_pose_index.end())
//                     poses_database[loop_second].linked_pose_index.push_back(loop_first);
//             }
//         }
//         else
//         {
//             printf("cannot find loop pose %d and %d, we have %d poses!\n", loop_first, loop_second, poses_database.size());
//         }
//     }

//     // if the current pose is new keyframe
//     bool is_new_keyframe;
//     if(this_pose_input->pose.covariance[0] > 0)
//         is_new_keyframe = true;
//     else
//         is_new_keyframe = false;
//     // the corner case that the first frame of the system
//     if(poses_database.size() == 0)
//         is_new_keyframe = true;
//     if(is_new_keyframe)
//     {
//         // add new pose
//         PoseElement this_pose_element;
//         int this_pose_index = poses_database.size();
//         this_pose_element.cam_pose = input_pose;
//         this_pose_element.loop_pose = input_pose;
//         this_pose_element.cam_stamp = this_pose_input->header.stamp;
//         if(poses_database.size() > 0)
//         {
//             int relative_index = this_pose_input->pose.covariance[1];
//             this_pose_element.linked_pose_index.push_back(relative_index);
//             poses_database[relative_index].linked_pose_index.push_back(this_pose_index);
//         }
//         poses_database.push_back(this_pose_element);
//         local_surfels_indexs.insert(this_pose_index);
//         // printf("add %d keyframe, with pose (%f, %f, %f, %f) and position (%f, %f, %f)!\n",
//         //     poses_database.size() - 1,
//         //     poses_database.back().cam_pose.orientation.x,
//         //     poses_database.back().cam_pose.orientation.y,
//         //     poses_database.back().cam_pose.orientation.z,
//         //     poses_database.back().cam_pose.orientation.w,
//         //     poses_database.back().cam_pose.position.x,
//         //     poses_database.back().cam_pose.position.y,
//         //     poses_database.back().cam_pose.position.z
//         // );
//     }

//     // push the msg into the buffer for fusion
//     int relative_index = this_pose_input->pose.covariance[1];
//     Eigen::Matrix4d reference_pose, fuse_pose, relative_pose;
//     pose_ros2eigen(poses_database[relative_index].cam_pose, reference_pose);
//     pose_ros2eigen(input_pose, fuse_pose);
//     relative_pose = reference_pose.inverse() * fuse_pose;
//     geometry_msgs::Pose relative_pose_ros;
//     pose_eigen2ros(relative_pose, relative_pose_ros);
//     pose_reference_buffer.push_back(std::make_tuple(loop_stamp_input->header.stamp, relative_pose_ros, relative_index));
//     synchronize_msgs();
// }

// bool SurfelMap::synchronize_buffer()
// {
//     if(!has_await_pose)
//         return false;
    
//     double pose_time = await_pose.header.stamp.toSec();
//     bool find_image = false;
//     bool find_depth = false;
//     bool pose_ahead = false;
//     for(int i =0; i < image_buffer.size(); i++)
//     {
//         double this_image_time = image_buffer.front().first.toSec();
//         if(fabs(this_image_time - pose_time) < 0.01)
//         {
//             find_image = true;
//             break;
//         }
//         else if(this_image_time < pose_time)
//         {
//             image_buffer.pop_front();
//         }
//         else
//         {
//             pose_ahead = true;
//         }
//     }
//     for(int i =0; i < depth_buffer.size(); i++)
//     {
//         double this_depth_time = depth_buffer.front().first.toSec();
//         if(fabs(this_depth_time - pose_time) < 0.01)
//         {
//             find_depth = true;
//             break;
//         }
//         else if(this_depth_time < pose_time)
//         {
//             depth_buffer.pop_front();
//         }
//         else
//         {
//             pose_ahead = true;
//         }
//     }
//     if(find_depth && find_image)
//     {
//         fuse_inputs();
//         image_buffer.pop_front();
//         depth_buffer.pop_front();
//         has_await_pose = false;
//         return true;
//     }
//     else if(pose_ahead)
//     {
//         has_await_pose = false;
//     }
//     return false;
// }

void SurfelMap::pose_ros2eigen(geometry_msgs::Pose &pose, Eigen::Matrix4d &T)
{
    T = Eigen::Matrix4d::Identity();
    Eigen::Quaterniond rotation_q;
    rotation_q.w() = pose.orientation.w;
    rotation_q.x() = pose.orientation.x;
    rotation_q.y() = pose.orientation.y;
    rotation_q.z() = pose.orientation.z;
    T.block<3,3>(0,0) = rotation_q.toRotationMatrix();
    T(0,3) = pose.position.x;
    T(1,3) = pose.position.y;
    T(2,3) = pose.position.z;
}

void SurfelMap::pose_eigen2ros(Eigen::Matrix4d &T, geometry_msgs::Pose &pose)
{
    Eigen::Quaterniond rotation_q(T.block<3,3>(0,0));
    pose.orientation.w = rotation_q.w();
    pose.orientation.x = rotation_q.x();
    pose.orientation.y = rotation_q.y();
    pose.orientation.z = rotation_q.z();
    pose.position.x = T(0,3);
    pose.position.y = T(1,3);
    pose.position.z = T(2,3);
}

// this is a naive implementation
// void SurfelMap::loop_stamp_input(const geometry_msgs::PointStampedConstPtr &loop_stamp)
// {
//     printf("receive loop info.\n");

//     double this_frame_stamp = loop_stamp->point.x;
//     double loop_frame_stamp = loop_stamp->point.y;
//     int this_frame_index = -1;
//     int loop_frame_index = -1;
//     for(int i = 0; i < poses_database.size(); i++)
//     {
//         double test_frame_stamp = poses_database[i].cam_stamp.toSec();
//         if(fabs(test_frame_stamp - this_frame_stamp) < 0.01)
//             this_frame_index = i;
//         if(fabs(test_frame_stamp - loop_frame_stamp) < 0.01)
//             loop_frame_index = i;
//     }
//     if(this_frame_index > 0 && loop_frame_index > 0)
//     {
//         poses_database[this_frame_index].linked_pose_index.push_back(loop_frame_index);
//         poses_database[loop_frame_index].linked_pose_index.push_back(this_frame_index);
//     }
//     else
//     {
//         printf("receive invaild loops!\n");
//         return;
//     }
//     printf("receive loop info %d <----> %d\n", loop_frame_index, this_frame_index);

//     // add local surfels into the local surfels
//     vector<int> new_driftless_poses;
//     get_driftfree_poses(poses_database.size() - 1, new_driftless_poses);
//     vector<int> poses_to_add;
//     for(int i = 0; i < new_driftless_poses.size(); i++)
//     {
//         if(local_surfels_indexs.find(new_driftless_poses[i]) == local_surfels_indexs.end())
//             poses_to_add.push_back(new_driftless_poses[i]);
//     }
//     if(poses_to_add.size() == 0)
//         return;
//     printf("loop! this pose %d, need to add!\n", poses_database.size() - 1);
//     for (auto a : poses_to_add)
//         std::cout << a << " ";

//     // add the surfels
//     // 1.0 add indexs
//     local_surfels_indexs.insert(poses_to_add.begin(), poses_to_add.end());

//     // 2.0 add surfels
//     // 2.1 remove the inactive_pointcloud
//     printf("\n");
//     std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
//     start_time = std::chrono::system_clock::now();

//     for(int add_i = 0; add_i < poses_to_add.size(); add_i++)
//     {
//         int add_index = poses_to_add[add_i];
//         int point_num = poses_database[add_index].attached_surfels.size();
//         int pre_size = inactive_pointcloud->size();

//         PointCloud::iterator begin_ptr;
//         PointCloud::iterator end_ptr;
//         begin_ptr = inactive_pointcloud->begin() + poses_database[add_index].points_begin_index;
//         end_ptr = inactive_pointcloud->begin() + poses_database[add_index].points_begin_index + poses_database[add_index].attached_surfels.size();
//         inactive_pointcloud->erase(begin_ptr, end_ptr);

//         for(int pi = poses_database[add_index].points_pose_index + 1; pi < pointcloud_pose_index.size(); pi++)
//         {
//             poses_database[pointcloud_pose_index[pi]].points_begin_index -= point_num;
//             poses_database[pointcloud_pose_index[pi]].points_pose_index -= 1; 
//         }
//         pointcloud_pose_index.erase(pointcloud_pose_index.begin() + poses_database[add_index].points_pose_index);
//         poses_database[add_index].points_pose_index = -1;
        
//         printf("erase %d points of pose %d, from %d -> %d.\n", point_num, add_index, pre_size, inactive_pointcloud->size());
//     }

//     // 2.3 add the surfels into local
//     for(int pi = 0; pi < poses_to_add.size(); pi++)
//     {
//         int pose_index = poses_to_add[pi];
//         local_surfels.insert(
//             local_surfels.end(),
//             poses_database[pose_index].attached_surfels.begin(),
//             poses_database[pose_index].attached_surfels.end());
//         poses_database[pose_index].attached_surfels.clear();
//         poses_database[pose_index].points_begin_index = -1;
//     }
//     end_time = std::chrono::system_clock::now();
//     std::chrono::duration<double> move_pointcloud_time = end_time - start_time;
//     printf("move surfels cost %f ms.\n", move_pointcloud_time.count()*1000.0);

//     // check the data
//     printf("check after!\n");
//     vector<std::pair<int, int>> pose_memeory_vector;
//     for(int i = 0; i < poses_database.size(); i++)
//     {
//         int begin_index = poses_database[i].points_begin_index;
//         int surfel_size = poses_database[i].attached_surfels.size();
//         if(begin_index >= 0)
//             pose_memeory_vector.push_back(std::make_pair(begin_index, surfel_size));
//     }
//     std::sort(
//         pose_memeory_vector.begin(),
//         pose_memeory_vector.end(),
//         []( const std::pair<int,int>& first, const std::pair<int,int>& second)
//         {
//             return first.first < second.first;
//         }
//     );
//     printf("we have %d pose pointclouds.\n", pose_memeory_vector.size());
//     int accumulate_index = 0;
//     for(int i = 0; i < pose_memeory_vector.size(); i++)
//     {
//         printf("cloud begin at %d have %d points.\n", pose_memeory_vector[i].first, pose_memeory_vector[i].second);
//         if(accumulate_index != pose_memeory_vector[i].first && pose_memeory_vector[i].second > 0)
//             printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!error 2!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
//         accumulate_index += pose_memeory_vector[i].second;
//     }
// }

// void SurfelMap::loop_stamp_input(const geometry_msgs::PointStampedConstPtr &loop_stamp)
// {
//     printf("receive loop info.\n");

//     double this_frame_stamp = loop_stamp->point.x;
//     double loop_frame_stamp = loop_stamp->point.y;
//     int this_frame_index = -1;
//     int loop_frame_index = -1;
//     for(int i = 0; i < poses_database.size(); i++)
//     {
//         double test_frame_stamp = poses_database[i].cam_stamp.toSec();
//         if(fabs(test_frame_stamp - this_frame_stamp) < 0.01)
//             this_frame_index = i;
//         if(fabs(test_frame_stamp - loop_frame_stamp) < 0.01)
//             loop_frame_index = i;
//     }
//     if(this_frame_index > 0 && loop_frame_index > 0)
//     {
//         poses_database[this_frame_index].linked_pose_index.push_back(loop_frame_index);
//         poses_database[loop_frame_index].linked_pose_index.push_back(this_frame_index);
//     }
//     else
//     {
//         printf("receive invaild loops!\n");
//         return;
//     }
//     printf("receive loop info %d <----> %d\n", loop_frame_index, this_frame_index);

//     // add local surfels into the local surfels
//     vector<int> new_driftless_poses;
//     get_driftfree_poses(poses_database.size() - 1, new_driftless_poses);
//     vector<int> poses_to_add;
//     for(int i = 0; i < new_driftless_poses.size(); i++)
//     {
//         if(local_surfels_indexs.find(new_driftless_poses[i]) == local_surfels_indexs.end())
//             poses_to_add.push_back(new_driftless_poses[i]);
//     }
//     if(poses_to_add.size() == 0)
//         return;
//     printf("loop! this pose %d, need to add!\n", poses_database.size() - 1);
//     for (auto a : poses_to_add)
//         std::cout << a << " ";

//     // add the surfels
//     // 1.0 add indexs
//     local_surfels_indexs.insert(poses_to_add.begin(), poses_to_add.end());

//     // 2.0 add surfels
//     // 2.1 remove the inactive_pointcloud
//     printf("\n");
//     std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
//     start_time = std::chrono::system_clock::now();
//     std::vector<std::pair<int, int>> remove_info;//first, pointcloud start, pointcloud size, pointcloud pose index
//     for(int add_i = 0; add_i < poses_to_add.size(); add_i++)
//     {
//         int add_index = poses_to_add[add_i];
//         int pointcloud_pose_index = poses_database[add_index].points_pose_index;
//         remove_info.push_back(std::make_pair(pointcloud_pose_index, add_index));
//     }
//     std::sort(
//     remove_info.begin(),
//     remove_info.end(),
//     []( const std::pair<int, int >& first, const std::pair<int, int>& second)
//     {
//         return first.first < second.first;
//     }
//     );
//     // printf("sort done!\n");
//     // for(int i = 0; i < remove_info.size(); i++)
//     // {
//     //     printf("%d, %d\n", remove_info[i].first, remove_info[i].second);
//     // }
//     int remove_begin_index = remove_info[0].second;
//     int remove_points_size = poses_database[remove_begin_index].attached_surfels.size();
//     int remove_pose_size = 1;
//     for(int remove_i = 1; remove_i <= remove_info.size(); remove_i++)
//     {
//         bool need_remove = false;
//         if(remove_i == remove_info.size())
//             need_remove = true;
//         if(remove_i < remove_info.size())
//         {
//             if(remove_info[remove_i].first != (remove_info[remove_i-1].first + 1))
//                 need_remove = true;
//         }
//         if(!need_remove)
//         {
//             int this_pose_index = remove_info[remove_i].second;
//             remove_points_size += poses_database[this_pose_index].attached_surfels.size();
//             remove_pose_size += 1;
//             continue;
//         }

//         int remove_end_index = remove_info[remove_i - 1].second;
//         printf("remove from pose %d -> %d, has %d points\n", remove_begin_index, remove_end_index, remove_points_size);

//         PointCloud::iterator begin_ptr;
//         PointCloud::iterator end_ptr;
//         begin_ptr = inactive_pointcloud->begin() + poses_database[remove_begin_index].points_begin_index;
//         end_ptr = begin_ptr + remove_points_size;
//         inactive_pointcloud->erase(begin_ptr, end_ptr);
        
//         for(int pi = poses_database[remove_end_index].points_pose_index + 1; pi < pointcloud_pose_index.size(); pi++)
//         {
//             poses_database[pointcloud_pose_index[pi]].points_begin_index -= remove_points_size;
//             poses_database[pointcloud_pose_index[pi]].points_pose_index -= remove_pose_size; 
//         }
 
//         pointcloud_pose_index.erase(
//             pointcloud_pose_index.begin() + poses_database[remove_begin_index].points_pose_index,
//             pointcloud_pose_index.begin() + poses_database[remove_end_index].points_pose_index + 1
//         );


//         if(remove_i < remove_info.size())
//         {
//             remove_begin_index = remove_info[remove_i].second;;
//             remove_points_size = poses_database[remove_begin_index].attached_surfels.size();
//             remove_pose_size = 1;
//         }
//     }

//     // 2.3 add the surfels into local
//     for(int pi = 0; pi < poses_to_add.size(); pi++)
//     {
//         int pose_index = poses_to_add[pi];
//         local_surfels.insert(
//             local_surfels.end(),
//             poses_database[pose_index].attached_surfels.begin(),
//             poses_database[pose_index].attached_surfels.end());
//         poses_database[pose_index].attached_surfels.clear();
//         poses_database[pose_index].points_begin_index = -1;
//         poses_database[pose_index].points_pose_index = -1;
//     }
//     end_time = std::chrono::system_clock::now();
//     std::chrono::duration<double> move_pointcloud_time = end_time - start_time;
//     printf("move surfels cost %f ms.\n", move_pointcloud_time.count()*1000.0);

//     // // check the data
//     // printf("check after!\n");
//     // vector<std::pair<int, int>> pose_memeory_vector;
//     // for(int i = 0; i < poses_database.size(); i++)
//     // {
//     //     int begin_index = poses_database[i].points_begin_index;
//     //     int surfel_size = poses_database[i].attached_surfels.size();
//     //     if(begin_index >= 0)
//     //         pose_memeory_vector.push_back(std::make_pair(begin_index, surfel_size));
//     // }
//     // std::sort(
//     //     pose_memeory_vector.begin(),
//     //     pose_memeory_vector.end(),
//     //     []( const std::pair<int,int>& first, const std::pair<int,int>& second)
//     //     {
//     //         return first.first < second.first;
//     //     }
//     // );
//     // printf("we have %d pose pointclouds.\n", pose_memeory_vector.size());
//     // int accumulate_index = 0;
//     // for(int i = 0; i < pose_memeory_vector.size(); i++)
//     // {
//     //     printf("cloud begin at %d have %d points.\n", pose_memeory_vector[i].first, pose_memeory_vector[i].second);
//     //     if(accumulate_index != pose_memeory_vector[i].first && pose_memeory_vector[i].second > 0)
//     //         printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!error 2!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
//     //     accumulate_index += pose_memeory_vector[i].second;
//     // }
// }

void SurfelMap::warp_inactive_surfels_cpu_kernel(int thread_i, int thread_num)
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    start_time = std::chrono::system_clock::now();
    int step = poses_database.size() / thread_num;
    int begin_index = step * thread_i;
    int end_index = begin_index + step;
    if (thread_i == thread_num - 1)
        end_index = poses_database.size();

    for(int i = begin_index; i < end_index; i ++)
    {
        if( poses_database[i].cam_pose.position.x == poses_database[i].loop_pose.position.x &&
            poses_database[i].cam_pose.position.y == poses_database[i].loop_pose.position.y &&
            poses_database[i].cam_pose.position.z == poses_database[i].loop_pose.position.z
            )
            continue;
        if(poses_database[i].attached_surfels.size() == 0)
        {
            poses_database[i].cam_pose = poses_database[i].loop_pose;
            continue;
        }

        PointCloud::Ptr warped_new_pointcloud(new PointCloud);

        Eigen::Matrix4d pre_pose, after_pose;
        Eigen::Matrix4f warp_matrix;
        pose_ros2eigen(poses_database[i].cam_pose, pre_pose);
        pose_ros2eigen(poses_database[i].loop_pose, after_pose);
        warp_matrix = (after_pose * pre_pose.inverse()).cast<float>();
        Eigen::MatrixXf point_positions(4, poses_database[i].attached_surfels.size());
        Eigen::MatrixXf point_norms(3, poses_database[i].attached_surfels.size());
        for(int surfel_i = 0; surfel_i < poses_database[i].attached_surfels.size(); surfel_i++)
        {
            point_positions(0,surfel_i) = poses_database[i].attached_surfels[surfel_i].px;
            point_positions(1,surfel_i) = poses_database[i].attached_surfels[surfel_i].py;
            point_positions(2,surfel_i) = poses_database[i].attached_surfels[surfel_i].pz;
            point_positions(3,surfel_i) = 1.0;
            point_norms(0,surfel_i) = poses_database[i].attached_surfels[surfel_i].nx;
            point_norms(1,surfel_i) = poses_database[i].attached_surfels[surfel_i].ny;
            point_norms(2,surfel_i) = poses_database[i].attached_surfels[surfel_i].nz;
        }
        point_positions = warp_matrix * point_positions;
        point_norms = warp_matrix.block<3,3>(0,0) * point_norms;
        for(int surfel_i = 0; surfel_i < poses_database[i].attached_surfels.size(); surfel_i++)
        {
            poses_database[i].attached_surfels[surfel_i].px = point_positions(0,surfel_i);
            poses_database[i].attached_surfels[surfel_i].py = point_positions(1,surfel_i);
            poses_database[i].attached_surfels[surfel_i].pz = point_positions(2,surfel_i);
            poses_database[i].attached_surfels[surfel_i].nx = point_norms(0,surfel_i);
            poses_database[i].attached_surfels[surfel_i].ny = point_norms(1,surfel_i);
            poses_database[i].attached_surfels[surfel_i].nz = point_norms(2,surfel_i);

            PointType new_point;
            new_point.x = poses_database[i].attached_surfels[surfel_i].px;
            new_point.y = poses_database[i].attached_surfels[surfel_i].py;
            new_point.z = poses_database[i].attached_surfels[surfel_i].pz;
            new_point.intensity = poses_database[i].attached_surfels[surfel_i].color;
            warped_new_pointcloud->push_back(new_point);
        }
        poses_database[i].cam_pose = poses_database[i].loop_pose;
        std::copy(&warped_new_pointcloud->front(), &warped_new_pointcloud->back(), &inactive_pointcloud->at(poses_database[i].points_begin_index));
    }
    end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> used_time = end_time - start_time;
    double all_time = used_time.count() * 1000.0;
    printf("warp kernel %d, cost %f ms.\n", thread_i, all_time);
}

void SurfelMap::warp_active_surfels_cpu_kernel(int thread_i, int thread_num, Eigen::Matrix4f transform_m)
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    start_time = std::chrono::system_clock::now();
    int step = local_surfels.size() / thread_num;
    int begin_index = step * thread_i;
    int end_index = begin_index + step;
    if (thread_i == thread_num - 1)
        end_index = local_surfels.size();
    int surfel_num = end_index - begin_index;

    Eigen::MatrixXf point_positions(4, surfel_num);
    Eigen::MatrixXf point_norms(3, surfel_num);
    for(int i = 0; i < surfel_num; i++)
    {
        point_positions(0, i) = local_surfels[i + begin_index].px;
        point_positions(1, i) = local_surfels[i + begin_index].py;
        point_positions(2, i) = local_surfels[i + begin_index].pz;
        point_positions(3, i) = 1.0;
        point_norms(0, i) = local_surfels[i + begin_index].nx;
        point_norms(1, i) = local_surfels[i + begin_index].ny;
        point_norms(2, i) = local_surfels[i + begin_index].nz;
    }
    point_positions = transform_m * point_positions;
    point_norms = transform_m.block<3,3>(0,0) * point_norms;
    for(int i = 0; i < surfel_num; i++)
    {
        local_surfels[i + begin_index].px = point_positions(0, i);
        local_surfels[i + begin_index].py = point_positions(1, i);
        local_surfels[i + begin_index].pz = point_positions(2, i);
        local_surfels[i + begin_index].nx = point_norms(0, i);
        local_surfels[i + begin_index].ny = point_norms(1, i);
        local_surfels[i + begin_index].nz = point_norms(2, i);
    }

    end_time = std::chrono::system_clock::now();
    std::chrono::duration<double> used_time = end_time - start_time;
    double all_time = used_time.count() * 1000.0;
    // printf("warp kernel %d, cost %f ms.\n", thread_i, all_time);
}

void SurfelMap::warp_surfels()
{
    warp_thread_pool.clear();
    warp_thread_num = 10;
    // warp inactive surfels
    for(int i = 0; i < warp_thread_num; i++)
    {
        std::thread this_thread(&SurfelMap::warp_inactive_surfels_cpu_kernel, this, i, warp_thread_num);
        warp_thread_pool.push_back(std::move(this_thread));
    }

    // for(int i = 0; i < warp_thread_pool.size(); i++)
    //     if(warp_thread_pool[i].joinable())
    //         warp_thread_pool[i].join();
    // warp_thread_pool.clear();

    // warp active surfels
    int local_index = *local_surfels_indexs.begin();
    Eigen::Matrix4d pre_pose, loop_pose;
    Eigen::Matrix4f warp_pose;
    pose_ros2eigen(poses_database[local_index].cam_pose, pre_pose);
    pose_ros2eigen(poses_database[local_index].loop_pose, loop_pose);
    warp_pose = (loop_pose * pre_pose.inverse()).cast<float>();
    
    for(int i = 0; i < warp_thread_num; i++)
    {
        std::thread this_thread(&SurfelMap::warp_active_surfels_cpu_kernel, this, i, warp_thread_num, warp_pose);
        warp_thread_pool.push_back(std::move(this_thread));
    }

    for(int i = 0; i < warp_thread_pool.size(); i++)
        if(warp_thread_pool[i].joinable())
            warp_thread_pool[i].join();
}

// void SurfelMap::warp_inactive_surfels()
// {
//     std::chrono::time_point<std::chrono::system_clock> all_start_time;
//     std::chrono::time_point<std::chrono::system_clock> all_end_time;
//     all_start_time = std::chrono::system_clock::now();
    
//     std::vector<std::thread> thread_pool;
//     int thread_num = 10;
//     for(int i = 0; i < thread_num; i++)
//     {
//         std::thread this_thread(&SurfelMap::warp_inactive_surfels_cpu_kernel, this, i, thread_num);
//         thread_pool.push_back(std::move(this_thread));
//     }
//     for(int i = 0; i < thread_num; i++)
//         if(thread_pool[i].joinable())
//             thread_pool[i].join();

//     all_end_time = std::chrono::system_clock::now();
//     std::chrono::duration<double> all_used_time = all_end_time - all_start_time;
//     printf("warp total cost %f ms.\n", (all_used_time.count()*1000.0) );
// }

// void SurfelMap::loop_path_input(const nav_msgs::PathConstPtr &loop_path_input)
// {
//     printf("\n\n\nreceive loop path!\n");
//     int received_pose_num = loop_path_input->poses.size();
//     bool loop_changed = false;
//     for(int i = 0; i < poses_database.size(); i++)
//     {
//         poses_database[i].loop_pose = loop_path_input->poses[i].pose;
//         if( loop_path_input->poses[i].pose.position.x != poses_database[i].cam_pose.position.x
//             || loop_path_input->poses[i].pose.position.y != poses_database[i].cam_pose.position.y
//             || loop_path_input->poses[i].pose.position.z != poses_database[i].cam_pose.position.z)
//             loop_changed = true;
//     }

//     // for local surfels warp
//     if( poses_database.size() > 0)
//     {
//         if(poses_database.back().cam_pose.position.x != poses_database.back().loop_pose.position.x
//             ||  poses_database.back().cam_pose.position.y != poses_database.back().loop_pose.position.y
//             ||  poses_database.back().cam_pose.position.z != poses_database.back().loop_pose.position.z)
//         {
//             Eigen::Matrix4d pre_pose, after_pose;
//             pose_ros2eigen(poses_database.back().cam_pose, pre_pose);
//             pose_ros2eigen(poses_database.back().loop_pose, after_pose);
//             local_loop_warp = after_pose * pre_pose.inverse() * local_loop_warp;
//             std::cout << "the warp matrix is :\n" << local_loop_warp << endl;
//         }
//     }

//     // TO DO
//     if(loop_changed)
//     {
//         printf("the loop changed! warp the inactive surfels!!\n");
//         warp_inactive_surfels();
//     }

//     // put into new pose
//     if(loop_path_input->poses.size() > poses_database.size())
//     {
//         await_pose = loop_path_input->poses[poses_database.size()];
//         has_await_pose = true;
//     }

//     // fuse or initialize surfels
//     synchronize_buffer();
// }

void SurfelMap::calculate_memory_usage()
{
    double usgae_KB = 0;
    usgae_KB += local_surfels.size() * sizeof(SurfelElement)  / 1024.0;
    usgae_KB += poses_database.size() * sizeof(PoseElement) / 1024.0;
    usgae_KB += local_surfels_indexs.size() * sizeof(int) / 1024.0;
    // usgae_KB += inactive_pointcloud->size() * sizeof(PointType) / 1024.0;
    usgae_KB += inactive_pointcloud->size() * sizeof(SurfelElement)  / 1024.0;
    printf("the process comsumes %f KB\n", usgae_KB);
}
void SurfelMap::publish_pose_graph(ros::Time pub_stamp, int reference_index)
{
    nav_msgs::Path loop_path;
    loop_path.header.stamp = pub_stamp;
    loop_path.header.frame_id = "world";

    visualization_msgs::Marker loop_marker;
    loop_marker.header.frame_id = "world";
    loop_marker.header.stamp = pub_stamp;
    loop_marker.ns = "namespace";
    loop_marker.id = 0;
    loop_marker.type = visualization_msgs::Marker::LINE_LIST;
    loop_marker.action = visualization_msgs::Marker::ADD;
    loop_marker.scale.x = 0.01;
    loop_marker.scale.y = 0.01;
    loop_marker.scale.z = 0.01;
    loop_marker.color.a = 1.0; // Don't forget to set the alpha!
    loop_marker.color.r = 1.0;
    loop_marker.color.g = 0.0;
    loop_marker.color.b = 0.0;
    for(int i = 0; i < poses_database.size(); i++)
    {
        geometry_msgs::PoseStamped loop_pose;
        loop_pose.header.stamp = poses_database[i].cam_stamp;
        loop_pose.pose = poses_database[i].cam_pose;

        loop_path.poses.push_back(loop_pose);

        for(int j = 0; j < poses_database[i].linked_pose_index.size(); j++)
        {
            if(     poses_database[i].linked_pose_index[j] != i-1 
                &&  poses_database[i].linked_pose_index[j] != i+1
                &&  poses_database[i].linked_pose_index[j] > i
                )
            {
                geometry_msgs::Point one_point, another_point;
                one_point.x = poses_database[i].loop_pose.position.x;
                one_point.y = poses_database[i].loop_pose.position.y;
                one_point.z = poses_database[i].loop_pose.position.z;
                another_point.x = poses_database[poses_database[i].linked_pose_index[j]].loop_pose.position.x;
                another_point.y = poses_database[poses_database[i].linked_pose_index[j]].loop_pose.position.y;
                another_point.z = poses_database[poses_database[i].linked_pose_index[j]].loop_pose.position.z;
                loop_marker.points.push_back(one_point);
                loop_marker.points.push_back(another_point);
            }
        }
    }

    loop_path_publish.publish(loop_path);
    loop_marker_publish.publish(loop_marker);

    // publish driftfree poses
    visualization_msgs::Marker driftfree_marker;
    driftfree_marker.header.frame_id = "world";
    driftfree_marker.header.stamp = pub_stamp;
    driftfree_marker.ns = "namespace";
    driftfree_marker.id = 0;
    driftfree_marker.type = visualization_msgs::Marker::SPHERE_LIST;
    driftfree_marker.action = visualization_msgs::Marker::ADD;
    driftfree_marker.scale.x = 1.1;
    driftfree_marker.scale.y = 1.1;
    driftfree_marker.scale.z = 1.1;
    driftfree_marker.color.a = 1.0; // Don't forget to set the alpha!
    driftfree_marker.color.r = 1.0;
    driftfree_marker.color.g = 0.0;
    driftfree_marker.color.b = 0.0;
    vector<int> driftfree_indexs;
    get_driftfree_poses(reference_index, driftfree_indexs, drift_free_poses);
    for(int i = 0; i < driftfree_indexs.size(); i++)
    {
        geometry_msgs::Point one_point;
        one_point.x = poses_database[driftfree_indexs[i]].cam_pose.position.x;
        one_point.y = poses_database[driftfree_indexs[i]].cam_pose.position.y;
        one_point.z = poses_database[driftfree_indexs[i]].cam_pose.position.z;
        driftfree_marker.points.push_back(one_point);
    }
    driftfree_path_publish.publish(driftfree_marker);
}

// void SurfelMap::fuse_inputs()
// {
//     printf("fuse image, depth, pose!\n");
//     if (poses_database.size() == 0)
//         initialize_map(image, depth, cam_pose, cam_time);
//     else
//         fuse_map(image, depth, cam_pose, cam_time);
// }

// void SurfelMap::initialize_map(cv::Mat image, cv::Mat depth, geometry_msgs::Pose pose, ros::Time stamp)
// {
//     Timer initialize_timer("initialize");
//     PoseElement first_pose;
//     first_pose.cam_pose = pose;
//     first_pose.cam_stamp = stamp;
//     vector<SurfelElement> surfels;
//     cuda_function::initialize_surfel_map_with_superpixel(image, depth, pose, surfels, fuse_param_gpuptr);
//     initialize_timer.middle("gpu part");
//     for(int i = 0; i < surfels.size(); i++)
//     {
//         if(surfels[i].update_times != 0)
//         {
//             SurfelElement this_surfel = surfels[i];
//             local_surfels.push_back(this_surfel);
//         }
//     }
//     initialize_timer.middle("cpu part");
//     initialize_timer.end();
//     poses_database.push_back(first_pose);
//     local_surfels_indexs.insert(0);
// }

void SurfelMap::fuse_map(cv::Mat image, cv::Mat depth, Eigen::Matrix4f pose_input, int reference_index)
{
    printf("fuse surfels with reference index %d and %d surfels!\n", reference_index, local_surfels.size());    
    Timer fuse_timer("fusing");

    vector<SurfelElement> new_surfels;
    fusion_functions.fuse_initialize_map(
        reference_index,
        image,
        depth,
        pose_input,
        local_surfels,
        new_surfels
    );
    // cuda_function::fuse_initialize_map(
    //     reference_index,
    //     image,
    //     depth,
    //     pose_input,
    //     local_surfels,
    //     new_surfels,
    //     fuse_param_gpuptr);
    // local_loop_warp = Eigen::Matrix4d::Identity();
    fuse_timer.middle("gpu part");

    // get the deleted surfel index
    vector<int> deleted_index;
    for(int i = 0; i < local_surfels.size(); i++)
    {
        if(local_surfels[i].update_times == 0)
            deleted_index.push_back(i);
    }
    fuse_timer.middle("delete index");

    // add new initialized surfels
    int add_surfel_num = 0;
    for(int i = 0; i < new_surfels.size(); i++)
    {
        if(new_surfels[i].update_times != 0)
        {
            SurfelElement this_surfel = new_surfels[i];
            if(deleted_index.size() > 0)
            {
                local_surfels[deleted_index.back()] = this_surfel;
                deleted_index.pop_back();
            }
            else
                local_surfels.push_back(this_surfel);
            add_surfel_num += 1;
        }
    }
    // remove deleted surfels
    while(deleted_index.size() > 0)
    {
        local_surfels[deleted_index.back()] = local_surfels.back();
        deleted_index.pop_back();
        local_surfels.pop_back();
    }
    fuse_timer.middle("cpu part");
    printf("add %d surfels, we now have %d local surfels.\n", add_surfel_num, local_surfels.size());
    fuse_timer.end();
}

// void SurfelMap::fuse_map(cv::Mat image, cv::Mat depth, geometry_msgs::Pose pose_input, ros::Time stamp)
// {
//     printf("fuse surfels!\n");    
//     Timer fuse_timer("fusing");
    
//     // warp the local surfels into the looped pose
//     geometry_msgs::Pose warp_pose;
//     pose_eigen2ros(local_loop_warp, warp_pose);
//     vector<SurfelElement> new_surfels;
//     cuda_function::warp_fuse_initialize_map(
//         warp_pose,
//         poses_database.size(),
//         image,
//         depth,
//         pose_input,
//         local_surfels,
//         new_surfels,
//         fuse_param_gpuptr);
//     local_loop_warp = Eigen::Matrix4d::Identity();
//     fuse_timer.middle("gpu part");

//     // get the deleted surfel index
//     vector<int> deleted_index;
//     for(int i = 0; i < local_surfels.size(); i++)
//     {
//         if(local_surfels[i].update_times == 0)
//             deleted_index.push_back(i);
//     }
//     printf("we have %d deleted surfels.\n", deleted_index.size());
//     fuse_timer.middle("delete index");

//     // add new initialized surfels
//     int add_surfel_num = 0;
//     for(int i = 0; i < new_surfels.size(); i++)
//     {
//         if(new_surfels[i].update_times != 0)
//         {
//             SurfelElement this_surfel = new_surfels[i];
//             if(deleted_index.size() > 0)
//             {
//                 local_surfels[deleted_index.back()] = this_surfel;
//                 deleted_index.pop_back();
//             }
//             else
//                 local_surfels.push_back(this_surfel);
//             add_surfel_num += 1;
//         }
//     }
//     // remove deleted surfels
//     while(deleted_index.size() > 0)
//     {
//         local_surfels[deleted_index.back()] = local_surfels.back();
//         deleted_index.pop_back();
//         local_surfels.pop_back();
//     }
//     fuse_timer.middle("cpu part");
//     printf("add %d surfels, we now have %d local surfels.\n", add_surfel_num, local_surfels.size());
//     fuse_timer.end();

//     PoseElement this_pose_element;
//     int this_pose_index = poses_database.size();
//     int last_pose_index = poses_database.size() - 1;
//     this_pose_element.cam_pose = pose_input;
//     this_pose_element.loop_pose = pose_input;
//     this_pose_element.cam_stamp = stamp;
//     this_pose_element.linked_pose_index.push_back(last_pose_index);
//     poses_database.back().linked_pose_index.push_back(this_pose_index);
//     poses_database.push_back(this_pose_element);
//     local_surfels_indexs.insert(this_pose_index);
// }

void SurfelMap::publish_raw_pointcloud(cv::Mat &depth, cv::Mat &reference, geometry_msgs::Pose &pose)
{
    Eigen::Matrix3f rotation_R;
    Eigen::Vector3f translation_T;
    Eigen::Quaternionf rotation_q;
    rotation_q.w() = pose.orientation.w;
    rotation_q.x() = pose.orientation.x;
    rotation_q.y() = pose.orientation.y;
    rotation_q.z() = pose.orientation.z;
    rotation_R = rotation_q.toRotationMatrix();
    translation_T(0) = pose.position.x;
    translation_T(1) = pose.position.y;
    translation_T(2) = pose.position.z;

    PointCloud::Ptr pointcloud(new PointCloud);
    for(int i = 0; i < cam_width; i++)
    for(int j = 0; j < cam_height; j++)
    {
        float depth_value = depth.at<float>(j,i);
        Eigen::Vector3f cam_point;
        cam_point(0) = (i - cam_cx) * depth_value / cam_fx;
        cam_point(1) = (j - cam_cy) * depth_value / cam_fy;
        cam_point(2) = depth_value;
        Eigen::Vector3f world_point;
        world_point = rotation_R * cam_point + translation_T;

        PointType p;
        p.x = world_point(0);
        p.y = world_point(1);
        p.z = world_point(2);
        p.intensity = reference.at<uchar>(j,i);
        pointcloud->push_back(p);
    }
    pointcloud->header.frame_id = "world";
    raw_pointcloud_publish.publish(pointcloud);
    printf("publish raw point cloud with %d points.\n", pointcloud->size());
}

void SurfelMap::save_cloud(string save_path_name)
{
    printf("saving pointcloud ...\n");
    PointCloud::Ptr pointcloud(new PointCloud);
    for(int surfel_it = 0; surfel_it < local_surfels.size(); surfel_it++)
    {
        if(local_surfels[surfel_it].update_times < 5)
            continue;
        PointType p;
        p.x = local_surfels[surfel_it].px;
        p.y = local_surfels[surfel_it].py;
        p.z = local_surfels[surfel_it].pz;
        p.intensity = local_surfels[surfel_it].color;
        pointcloud->push_back(p);
    }
    
    (*pointcloud) += (*inactive_pointcloud);
    
    // pcl::io::savePLYFile(save_path_name.c_str(), *pointcloud);
    pcl::io::savePCDFile(save_path_name.c_str(), *pointcloud);
    printf("saving pointcloud done!\n");
}

void SurfelMap::push_a_surfel(vector<float> &vertexs, SurfelElement &this_surfel)
{
    int surfel_color = this_surfel.color;
    Eigen::Vector3f surfel_position;
    surfel_position(0) = this_surfel.px;
    surfel_position(1) = this_surfel.py;
    surfel_position(2) = this_surfel.pz;
    Eigen::Vector3f surfel_norm;
    surfel_norm(0) = this_surfel.nx;
    surfel_norm(1) = this_surfel.ny;
    surfel_norm(2) = this_surfel.nz;
    Eigen::Vector3f x_dir;
    x_dir(0) = -1 * this_surfel.ny;
    x_dir(1) = this_surfel.nx;
    x_dir(2) = 0;
    x_dir.normalize();
    Eigen::Vector3f y_dir;
    y_dir = surfel_norm.cross(x_dir);
    float radius = this_surfel.size;
    float h_r = radius * 0.5;
    float t_r = radius * 0.86603;
    Eigen::Vector3f point1, point2, point3, point4, point5, point6;
    point1 = surfel_position - x_dir * h_r - y_dir * t_r;
    point2 = surfel_position + x_dir * h_r - y_dir * t_r;
    point3 = surfel_position - x_dir * radius;
    point4 = surfel_position + x_dir * radius;
    point5 = surfel_position - x_dir * h_r + y_dir * t_r;
    point6 = surfel_position + x_dir * h_r + y_dir * t_r;
    vertexs.push_back(point1(0));vertexs.push_back(point1(1));vertexs.push_back(point1(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
    vertexs.push_back(point2(0));vertexs.push_back(point2(1));vertexs.push_back(point2(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
    vertexs.push_back(point3(0));vertexs.push_back(point3(1));vertexs.push_back(point3(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
    vertexs.push_back(point4(0));vertexs.push_back(point4(1));vertexs.push_back(point4(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
    vertexs.push_back(point5(0));vertexs.push_back(point5(1));vertexs.push_back(point5(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
    vertexs.push_back(point6(0));vertexs.push_back(point6(1));vertexs.push_back(point6(2));
    vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);vertexs.push_back(surfel_color);
}


void SurfelMap::save_mesh(string save_path_name)
{
    std::ofstream stream(save_path_name.c_str());
    if (!stream)
        return;
    std::vector<float> vertexs;
    for(int i = 0; i < poses_database.size(); i++)
    {
        for(int j = 0; j < poses_database[i].attached_surfels.size(); j++)
        {
            SurfelElement this_surfel = poses_database[i].attached_surfels[j];
            push_a_surfel(vertexs, this_surfel);
        }
    }

    for(int i = 0; i < local_surfels.size(); i++)
    {
        if(local_surfels[i].update_times < 5)
            continue;
        SurfelElement this_surfel = local_surfels[i];
        push_a_surfel(vertexs, this_surfel);
    }
    
    size_t numPoints = vertexs.size()/6;
    size_t numSurfels = numPoints/6;
    stream << "ply" << std::endl;
    stream << "format ascii 1.0" << std::endl;
    stream << "element vertex " << numPoints << std::endl;
    stream << "property float x" << std::endl;
    stream << "property float y" << std::endl;
    stream << "property float z" << std::endl;
    stream << "property uchar red" << std::endl;
    stream << "property uchar green" << std::endl;
    stream << "property uchar blue" << std::endl;
    stream << "element face " << numSurfels * 4 <<  std::endl;
    stream << "property list uchar int vertex_index" << std::endl;
    stream << "end_header" << std::endl;

    for(int i = 0; i < numPoints; i++)
    {
        for(int j = 0; j < 6; j++)
        {
            stream << vertexs[i*6+j] << " ";
        }
        stream << std::endl;
    }
    for(int i = 0; i < numSurfels; i++)
    {
        int p1, p2, p3, p4, p5, p6;
        p1 = i * 6 + 0;
        p2 = i * 6 + 1;
        p3 = i * 6 + 2;
        p4 = i * 6 + 3;
        p5 = i * 6 + 4;
        p6 = i * 6 + 5;
        stream << "3 " << p1 << " " << p2 << " " << p3 << std::endl;
        stream << "3 " << p2 << " " << p4 << " " << p3 << std::endl;
        stream << "3 " << p3 << " " << p4 << " " << p5 << std::endl;
        stream << "3 " << p5 << " " << p4 << " " << p6 << std::endl;
    }
    stream.close();
}


void SurfelMap::render_depth(geometry_msgs::Pose &pose)
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> total_time;
    start_time = std::chrono::system_clock::now();

    vector<float> positions;
    vector<float> normrs;
    for(int surfel_it = 0; surfel_it < local_surfels.size(); surfel_it++)
    {
        if(local_surfels[surfel_it].update_times < 5)
            continue;
        positions.push_back(local_surfels[surfel_it].px);
        positions.push_back(local_surfels[surfel_it].py);
        positions.push_back(local_surfels[surfel_it].pz);
        normrs.push_back(local_surfels[surfel_it].nx);
        normrs.push_back(local_surfels[surfel_it].ny);
        normrs.push_back(local_surfels[surfel_it].nz);
        normrs.push_back(local_surfels[surfel_it].size);
        // if(local_surfels[surfel_it].size != local_surfels[surfel_it].size)
        //     std::cout << "error!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << std::endl;
        // normrs.push_back(0.01);
    }

    end_time = std::chrono::system_clock::now();
    total_time = end_time - start_time;
    printf("render_depth: construct information vector cost %f ms.\n", total_time.count()*1000.0);
    start_time = std::chrono::system_clock::now();

    Eigen::Matrix4d eigen_pose;
    pose_ros2eigen(pose, eigen_pose);
    Eigen::Matrix4f eigen_pose_f = eigen_pose.cast<float>();
    vector<float> depth_results;
    render_tool.render_surfels(positions, normrs, depth_results, eigen_pose_f);

    end_time = std::chrono::system_clock::now();
    total_time = end_time - start_time;
    printf("render_depth: openGL render cost %f ms.\n", total_time.count()*1000.0);
    start_time = std::chrono::system_clock::now();

    cv::Mat depth_mat = cv::Mat(cam_height, cam_width, CV_32FC1);
    memcpy(depth_mat.data, depth_results.data(), depth_results.size()*sizeof(float));

    cv::Mat depth_uchar;
    depth_mat.convertTo(depth_uchar, CV_8UC1, 1.0/4.0*255.0, 0);
    cv::imshow("rendered depth", depth_uchar);
    cv::waitKey(10);
}

void SurfelMap::publish_neighbor_pointcloud(ros::Time pub_stamp, int reference_index)
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> total_time;
    start_time = std::chrono::system_clock::now();

    PointCloud::Ptr pointcloud(new PointCloud);
    pointcloud->reserve(local_surfels.size() + inactive_pointcloud->size());
    for(int surfel_it = 0; surfel_it < local_surfels.size(); surfel_it++)
    {
        if(local_surfels[surfel_it].update_times == 0)
            continue;
        PointType p;
        p.x = local_surfels[surfel_it].px;
        p.y = local_surfels[surfel_it].py;
        p.z = local_surfels[surfel_it].pz;
        p.intensity = local_surfels[surfel_it].color;
        pointcloud->push_back(p);
    }

    // add other pointcloud
    
    //METHOD 1, NAIVE ADD THE POINTS
    std::vector<int> neighbor_indexs;
    get_driftfree_poses(reference_index, neighbor_indexs, 2*drift_free_poses);
    for(int i = 0; i < neighbor_indexs.size(); i++)
    {
        int this_pose = neighbor_indexs[i];
        if(local_surfels_indexs.find(this_pose) != local_surfels_indexs.end())
            continue;
        int pointcloud_num = poses_database[this_pose].attached_surfels.size();
        int pointcloud_begin = poses_database[this_pose].points_begin_index;
        if(pointcloud_num <= 0)
            continue;
        pointcloud->insert(
            pointcloud->end(),
            inactive_pointcloud->begin()+pointcloud_begin,
            inactive_pointcloud->begin()+pointcloud_begin+pointcloud_num);
    }
    //NETHOD 1 ENDS

    // //METHOD 2, FIND THE SUCCESSIVELY MEMORY AND ADD
    // std::vector<int> neighbor_indexs;
    // get_driftfree_poses(reference_index, neighbor_indexs, 2*drift_free_poses);
    // std::vector<int> points_begin_end;
    // for(int i = 0; i < neighbor_indexs.size(); i++)
    // {
    //     int this_pose = neighbor_indexs[i];
    //     if(local_surfels_indexs.find(this_pose) != local_surfels_indexs.end())
    //         continue;
    //     int pointcloud_num = poses_database[this_pose].attached_surfels.size();
    //     int pointcloud_begin = poses_database[this_pose].points_begin_index;
    //     if(pointcloud_num <= 0)
    //         continue;
    //     points_begin_end.push_back(pointcloud_begin);
    //     points_begin_end.push_back(pointcloud_begin+pointcloud_num);
    // }
    // if(points_begin_end.size() > 0)
    // {
    //     std::sort(points_begin_end.begin(), points_begin_end.end());
    //     int points_add_begin = points_begin_end.front();
    //     bool need_to_add = false;
    //     for(int i = 0; i < points_begin_end.size() / 2; i++)
    //     {
    //         if(need_to_add)
    //             points_add_begin = points_begin_end[2*i];
    //         need_to_add = false;
    //         int this_end = points_begin_end[2*i+1];
    //         if(i == points_begin_end.size() / 2 - 1)
    //             need_to_add = true;
    //         else
    //         {
    //             int next_begin = points_begin_end[2*i + 2];
    //             if(next_begin != this_end + 1)
    //                 need_to_add = true;
    //         }
    //         if(need_to_add)
    //         {
    //             pointcloud->insert(
    //                 pointcloud->end(),
    //                 inactive_pointcloud->begin()+points_add_begin,
    //                 inactive_pointcloud->begin()+this_end);
    //         }
    //     }
    // }
    // //METHOD 2 ENDS

    end_time = std::chrono::system_clock::now();
    total_time = end_time - start_time;
    printf("construct point cloud cost %f ms.\n", total_time.count()*1000.0);
    start_time = std::chrono::system_clock::now();

    pointcloud->header.frame_id = "world";
    pcl_conversions::toPCL(pub_stamp, pointcloud->header.stamp);
    pointcloud_publish.publish(pointcloud);
    printf("publish point cloud with %d points, in active %d points.\n", pointcloud->size(), inactive_pointcloud->size());

    end_time = std::chrono::system_clock::now();
    total_time = end_time - start_time;
    printf("publish point cloud cost %f ms.\n", total_time.count()*1000.0);
}


void SurfelMap::publish_all_pointcloud(ros::Time pub_stamp)
{
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> total_time;
    start_time = std::chrono::system_clock::now();

    PointCloud::Ptr pointcloud(new PointCloud);
    pointcloud->reserve(local_surfels.size() + inactive_pointcloud->size());
    for(int surfel_it = 0; surfel_it < local_surfels.size(); surfel_it++)
    {
        if(local_surfels[surfel_it].update_times < 5)
            continue;
        PointType p;
        p.x = local_surfels[surfel_it].px;
        p.y = local_surfels[surfel_it].py;
        p.z = local_surfels[surfel_it].pz;
        p.intensity = local_surfels[surfel_it].color;
        pointcloud->push_back(p);
    }

    (*pointcloud) += (*inactive_pointcloud);

    end_time = std::chrono::system_clock::now();
    total_time = end_time - start_time;
    // printf("construct point cloud cost %f ms.\n", total_time.count()*1000.0);
    start_time = std::chrono::system_clock::now();

    pointcloud->header.frame_id = "world";
    pcl_conversions::toPCL(pub_stamp, pointcloud->header.stamp);
    pointcloud_publish.publish(pointcloud);
    printf("publish point cloud with %d points, inactive %d points.\n", pointcloud->size(), inactive_pointcloud->size());

    // end_time = std::chrono::system_clock::now();
    // total_time = end_time - start_time;
    // printf("publish point cloud cost %f ms.\n", total_time.count()*1000.0);
}

void SurfelMap::move_all_surfels()
{
    vector<int> poses_to_remove(local_surfels_indexs.begin(), local_surfels_indexs.end());
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> move_pointcloud_time;

    if(poses_to_remove.size() > 0)
    {
        
        start_time = std::chrono::system_clock::now();
        int added_surfel_num = 0;
        float sum_update_times = 0.0;
        for(int pi = 0; pi < poses_to_remove.size(); pi++)
        {
            int inactive_index = poses_to_remove[pi];
            poses_database[inactive_index].points_begin_index = inactive_pointcloud->size();
            poses_database[inactive_index].points_pose_index = pointcloud_pose_index.size();
            pointcloud_pose_index.push_back(inactive_index);
            for(int i = 0; i < local_surfels.size(); i++)
            {
                if(local_surfels[i].update_times > 0 && local_surfels[i].last_update == inactive_index)
                {
                    poses_database[inactive_index].attached_surfels.push_back(local_surfels[i]);

                    PointType p;
                    p.x = local_surfels[i].px;
                    p.y = local_surfels[i].py;
                    p.z = local_surfels[i].pz;
                    p.intensity = local_surfels[i].color;
                    inactive_pointcloud->push_back(p);

                    added_surfel_num += 1;
                    sum_update_times += local_surfels[i].update_times;

                    // delete the surfel from the local point
                    local_surfels[i].update_times = 0;
                }
            }
            // printf("remove pose %d from local poses, get %d surfels.\n", inactive_index, poses_database[inactive_index].attached_surfels.size());
            local_surfels_indexs.erase(inactive_index);
        }
        sum_update_times = sum_update_times / added_surfel_num;
        end_time = std::chrono::system_clock::now();
        move_pointcloud_time = end_time - start_time;
        printf("move surfels cost %f ms. the average update times is %f.\n", move_pointcloud_time.count()*1000.0, sum_update_times);
    }
}

void SurfelMap::move_add_surfels(int reference_index)
{
    // remove inactive surfels
    printf("get inactive surfels for pose %d.\n", reference_index);
    // vector<int> drift_poses;
    vector<int> poses_to_add;
    vector<int> poses_to_remove;
    get_add_remove_poses(reference_index, poses_to_add, poses_to_remove);
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;
    std::chrono::duration<double> move_pointcloud_time;

    if(poses_to_remove.size() > 0)
    {
        
        start_time = std::chrono::system_clock::now();
        int added_surfel_num = 0;
        float sum_update_times = 0.0;
        for(int pi = 0; pi < poses_to_remove.size(); pi++)
        {
            int inactive_index = poses_to_remove[pi];
            poses_database[inactive_index].points_begin_index = inactive_pointcloud->size();
            poses_database[inactive_index].points_pose_index = pointcloud_pose_index.size();
            pointcloud_pose_index.push_back(inactive_index);
            for(int i = 0; i < local_surfels.size(); i++)
            {
                if(local_surfels[i].update_times > 0 && local_surfels[i].last_update == inactive_index)
                {
                    poses_database[inactive_index].attached_surfels.push_back(local_surfels[i]);

                    PointType p;
                    p.x = local_surfels[i].px;
                    p.y = local_surfels[i].py;
                    p.z = local_surfels[i].pz;
                    p.intensity = local_surfels[i].color;
                    inactive_pointcloud->push_back(p);

                    added_surfel_num += 1;
                    sum_update_times += local_surfels[i].update_times;

                    // delete the surfel from the local point
                    local_surfels[i].update_times = 0;
                }
            }
            printf("remove pose %d from local poses, get %d surfels.\n", inactive_index, poses_database[inactive_index].attached_surfels.size());
            local_surfels_indexs.erase(inactive_index);
        }
        sum_update_times = sum_update_times / added_surfel_num;
        end_time = std::chrono::system_clock::now();
        move_pointcloud_time = end_time - start_time;
        printf("move surfels cost %f ms. the average update times is %f.\n", move_pointcloud_time.count()*1000.0, sum_update_times);
    }
    if(poses_to_add.size() > 0)
    {
        // 1.0 add indexs
        local_surfels_indexs.insert(poses_to_add.begin(), poses_to_add.end());
        // 2.0 add surfels
        // 2.1 remove from inactive_pointcloud
        start_time = std::chrono::system_clock::now();
        std::vector<std::pair<int, int>> remove_info;//first, pointcloud start, pointcloud size, pointcloud pose index
        for(int add_i = 0; add_i < poses_to_add.size(); add_i++)
        {
            int add_index = poses_to_add[add_i];
            int pointcloud_pose_index = poses_database[add_index].points_pose_index;
            remove_info.push_back(std::make_pair(pointcloud_pose_index, add_index));
        }
        std::sort(
        remove_info.begin(),
        remove_info.end(),
        []( const std::pair<int, int >& first, const std::pair<int, int>& second)
        {
            return first.first < second.first;
        }
        );
        int remove_begin_index = remove_info[0].second;
        int remove_points_size = poses_database[remove_begin_index].attached_surfels.size();
        int remove_pose_size = 1;
        for(int remove_i = 1; remove_i <= remove_info.size(); remove_i++)
        {
            bool need_remove = false;
            if(remove_i == remove_info.size())
                need_remove = true;
            if(remove_i < remove_info.size())
            {
                if(remove_info[remove_i].first != (remove_info[remove_i-1].first + 1))
                    need_remove = true;
            }
            if(!need_remove)
            {
                int this_pose_index = remove_info[remove_i].second;
                remove_points_size += poses_database[this_pose_index].attached_surfels.size();
                remove_pose_size += 1;
                continue;
            }

            int remove_end_index = remove_info[remove_i - 1].second;
            printf("remove from pose %d -> %d, has %d points\n", remove_begin_index, remove_end_index, remove_points_size);

            PointCloud::iterator begin_ptr;
            PointCloud::iterator end_ptr;
            begin_ptr = inactive_pointcloud->begin() + poses_database[remove_begin_index].points_begin_index;
            end_ptr = begin_ptr + remove_points_size;
            inactive_pointcloud->erase(begin_ptr, end_ptr);
            
            for(int pi = poses_database[remove_end_index].points_pose_index + 1; pi < pointcloud_pose_index.size(); pi++)
            {
                poses_database[pointcloud_pose_index[pi]].points_begin_index -= remove_points_size;
                poses_database[pointcloud_pose_index[pi]].points_pose_index -= remove_pose_size; 
            }
    
            pointcloud_pose_index.erase(
                pointcloud_pose_index.begin() + poses_database[remove_begin_index].points_pose_index,
                pointcloud_pose_index.begin() + poses_database[remove_end_index].points_pose_index + 1
            );


            if(remove_i < remove_info.size())
            {
                remove_begin_index = remove_info[remove_i].second;;
                remove_points_size = poses_database[remove_begin_index].attached_surfels.size();
                remove_pose_size = 1;
            }
        }

        // 2.3 add the surfels into local
        for(int pi = 0; pi < poses_to_add.size(); pi++)
        {
            int pose_index = poses_to_add[pi];
            local_surfels.insert(
                local_surfels.end(),
                poses_database[pose_index].attached_surfels.begin(),
                poses_database[pose_index].attached_surfels.end());
            poses_database[pose_index].attached_surfels.clear();
            poses_database[pose_index].points_begin_index = -1;
            poses_database[pose_index].points_pose_index = -1;
        }
        end_time = std::chrono::system_clock::now();
        move_pointcloud_time = end_time - start_time;
        printf("add surfels cost %f ms.\n", move_pointcloud_time.count()*1000.0);
    }
}

// void SurfelMap::get_inactive_surfels()
// {
//     printf("get inactive surfels!\n");
//     vector<int> drift_poses;
//     get_drift_poses(poses_database.size()-1, drift_poses);
//     if(drift_poses.size() == 0)
//         return;
//     int added_surfel_num = 0;
//     float sum_update_times = 0.0;
//     for(int pi = 0; pi < drift_poses.size(); pi++)
//     {
//         int inactive_index = drift_poses[pi];
//         poses_database[inactive_index].points_begin_index = inactive_pointcloud->size();
//         poses_database[inactive_index].points_pose_index = pointcloud_pose_index.size();
//         pointcloud_pose_index.push_back(inactive_index);
//         for(int i = 0; i < local_surfels.size(); i++)
//         {
//             if(local_surfels[i].update_times > 0 && local_surfels[i].last_update == inactive_index)
//             {
//                 poses_database[inactive_index].attached_surfels.push_back(local_surfels[i]);

//                 PointType p;
//                 p.x = local_surfels[i].px;
//                 p.y = local_surfels[i].py;
//                 p.z = local_surfels[i].pz;
//                 p.intensity = local_surfels[i].color;
//                 inactive_pointcloud->push_back(p);

//                 added_surfel_num += 1;
//                 sum_update_times += local_surfels[i].update_times;

//                 // delete the surfel from the local point
//                 local_surfels[i].update_times = 0;
//             }
//         }
//         printf("remove pose %d from local poses, get %d surfels.\n", inactive_index, poses_database[inactive_index].attached_surfels.size());
//         local_surfels_indexs.erase(inactive_index);
//     }
//     sum_update_times = sum_update_times / added_surfel_num;
// }

void SurfelMap::get_add_remove_poses(int root_index, vector<int> &pose_to_add, vector<int> &pose_to_remove)
{
    vector<int> driftfree_poses;
    get_driftfree_poses(root_index, driftfree_poses, drift_free_poses);
    {
        /*printf("\ndriftfree poses: ");
        for(int i = 0; i < driftfree_poses.size(); i++)
        {
            printf("%d, ", driftfree_poses[i]);
        }*/
    }
    pose_to_add.clear();
    pose_to_remove.clear();
    // get to add
    for(int i = 0; i < driftfree_poses.size(); i++)
    {
        int temp_pose = driftfree_poses[i];
        if(local_surfels_indexs.find(temp_pose) == local_surfels_indexs.end())
            pose_to_add.push_back(temp_pose);
    }
    {
        printf("\nto add: ");
        for(int i = 0; i < pose_to_add.size(); i++)
        {
            printf("%d, ", pose_to_add[i]);
        }
    }
    // get to remove
    for(auto i = local_surfels_indexs.begin(); i != local_surfels_indexs.end(); i++)
    {
        int temp_pose = *i;
        if( std::find(driftfree_poses.begin(), driftfree_poses.end(), temp_pose) ==  driftfree_poses.end() )
        {
            pose_to_remove.push_back(temp_pose);
        }
    }
    {
        printf("\nto remove: ");
        for(int i = 0; i < pose_to_remove.size(); i++)
        {
            printf("%d, ", pose_to_remove[i]);
        }
        printf("\n");
    }
}

void SurfelMap::get_driftfree_poses(int root_index, vector<int> &driftfree_poses, int driftfree_range)
{
    if(poses_database.size() < root_index + 1)
    {
        printf("get_driftfree_poses: pose database do not have the root index! This should only happen in initializaion!\n");
        return;
    }
    vector<int> this_level;
    vector<int> next_level;
    this_level.push_back(root_index);
    driftfree_poses.push_back(root_index);
    // get the drift
    for(int i = 1; i < driftfree_range; i++)
    {
        for(auto this_it = this_level.begin(); this_it != this_level.end(); this_it++)
        {
            for(auto linked_it = poses_database[*this_it].linked_pose_index.begin(); 
                linked_it != poses_database[*this_it].linked_pose_index.end();
                linked_it++)
            {
                bool already_saved = (find(driftfree_poses.begin(), driftfree_poses.end(), *linked_it) != driftfree_poses.end());
                if(!already_saved)
                {
                    next_level.push_back(*linked_it);
                    driftfree_poses.push_back(*linked_it);
                }
            }
        }
        this_level.swap(next_level);
        next_level.clear();
    }
}

// void SurfelMap::get_drift_poses(int root_index, vector<int> &drift_poses)
// {
//     if(poses_database.size() < root_index + 1)
//     {
//         printf("get_drift_poses: pose database do not have the root index! This should only happen in initializaion!\n");
//         return;
//     }
//     vector<int> vistited_poses;
//     drift_poses.clear();
//     vector<int> this_level;
//     vector<int> next_level;
//     this_level.push_back(root_index);
//     vistited_poses.push_back(root_index);
//     // get the drift
//     for(int i = 1; i < drift_free_poses; i++)
//     {
//         for(auto this_it = this_level.begin(); this_it != this_level.end(); this_it++)
//         {
//             for(auto linked_it = poses_database[*this_it].linked_pose_index.begin(); 
//                 linked_it != poses_database[*this_it].linked_pose_index.end();
//                 linked_it++)
//             {
//                 bool already_saved = (find(vistited_poses.begin(), vistited_poses.end(), *linked_it) != vistited_poses.end());
//                 if(!already_saved)
//                 {
//                     next_level.push_back(*linked_it);
//                     vistited_poses.push_back(*linked_it);
//                 }
//             }
//         }
//         this_level.swap(next_level);
//         next_level.clear();
//     }
//     // get the drift poses
//     for(auto this_it = this_level.begin(); this_it != this_level.end(); this_it++)
//     {
//         for(auto linked_it = poses_database[*this_it].linked_pose_index.begin(); 
//             linked_it != poses_database[*this_it].linked_pose_index.end();
//             linked_it++)
//         {
//             bool already_saved = (find(vistited_poses.begin(), vistited_poses.end(), *linked_it) != vistited_poses.end())
//                 || (find(drift_poses.begin(), drift_poses.end(), *linked_it) != drift_poses.end());
//             if(!already_saved)
//             {
//                 drift_poses.push_back(*linked_it);
//             }
//         }
//     }
// }

// void SurfelMap::fuse_one_frame(cv::Mat image, cv::Mat depth, geometry_msgs::Pose pose_input, ros::Time stamp)
// {
    // Timer fuse_timer("fusing");

    // // cuda fusion
    // cuda_function::fuse_surfel_map(
    //     poses_database.size(),
    //     image, depth, pose_input,
    //     local_surfels, fuse_param_gpuptr);
    // fuse_timer.middle("gpu part");

    // // get the deleted surfel index
    // vector<int> deleted_index;
    // for(int i = 0; i < local_surfels.size(); i++)
    // {
    //     if(local_surfels[i].update_times == 0)
    //         deleted_index.push_back(i);
    // }
    // printf("we have %d deleted surfels.\n", deleted_index.size());
    // fuse_timer.middle("delete index");
    
    // // rended successfully fused surfels
    // // first render the index map
    // int surfel_num = local_surfels.size();
    // vector<float> location_vector;
    // location_vector.resize(surfel_num*3);
    // vector<float> normr_vector;
    // normr_vector.resize(surfel_num*4);
    // int buffer_size = 0;
    // for(int i = 0; i < surfel_num; i++)
    // {
    //     if(local_surfels[i].last_update == poses_database.size())
    //     {
    //         location_vector[buffer_size * 3] = local_surfels[i].px;
    //         location_vector[buffer_size * 3 + 1] = local_surfels[i].py;
    //         location_vector[buffer_size * 3 + 2] = local_surfels[i].pz;
    //         normr_vector[buffer_size * 4] = local_surfels[i].nx;
    //         normr_vector[buffer_size * 4 + 1] = local_surfels[i].ny;
    //         normr_vector[buffer_size * 4 + 2] = local_surfels[i].nz;
    //         normr_vector[buffer_size * 4 + 3] = local_surfels[i].size;
    //         buffer_size++;
    //     }
    // }
    // location_vector.resize(buffer_size * 3);
    // normr_vector.resize(buffer_size * 4);
    // fuse_timer.middle("prepare the vectors");
    // vector<float> result_vector;
    // result_vector.resize(cam_width*cam_height);
    // Eigen::Matrix4f cam_in_world = Eigen::Matrix4f::Identity();
    // Eigen::Quaternionf rotation_q;
    // rotation_q.w() = pose_input.orientation.w;
    // rotation_q.x() = pose_input.orientation.x;
    // rotation_q.y() = pose_input.orientation.y;
    // rotation_q.z() = pose_input.orientation.z;
    // cam_in_world(0, 3) = pose_input.position.x;
    // cam_in_world(1, 3) = pose_input.position.y;
    // cam_in_world(2, 3) = pose_input.position.z;
    // cam_in_world.block(0, 0, 3, 3) = rotation_q.toRotationMatrix();
    // render_tool.render_surfels(location_vector, normr_vector, result_vector, cam_in_world);
    // fuse_timer.middle("render surfels");

    // // check the rendered depth map
    // cv::Mat depth_image = cv::Mat(cam_height, cam_width, CV_32FC1);
    // memcpy(depth_image.data, result_vector.data(), result_vector.size()*sizeof(float));
    // cv::Mat depth_uchar;
    // depth_image.convertTo(depth_uchar, CV_8U, 1.0/5.0*255.0, 0);
    // cv::Mat show_image;
    // cv::applyColorMap(depth_uchar, show_image, cv::COLORMAP_JET);
    // cv::imshow("rendered depth", show_image);
    // cv::waitKey(10);

    // // initialize left surfels
    // vector<SurfelElement> new_surfels;
    // new_surfels.resize(cam_width*cam_height);
    // cuda_function::initialize_surfel_map(poses_database.size(), image, depth, pose_input, new_surfels, fuse_param_gpuptr, result_vector);
    // int add_surfel_num = 0;
    // for(int i = 0; i < cam_width*cam_height; i++)
    // {
    //     if(new_surfels[i].update_times != 0 && new_surfels[i].size > MIN_SURFEL_SIZE)
    //     // if(new_surfels[i].update_times != 0)
    //     {
    //         SurfelElement this_surfel = new_surfels[i];
    //         if(deleted_index.size() > 0)
    //         {
    //             local_surfels[deleted_index.back()] = this_surfel;
    //             deleted_index.pop_back();
    //         }
    //         else
    //             local_surfels.push_back(this_surfel);
    //         add_surfel_num += 1;
    //     }
    // }
    // fuse_timer.middle("cpu part");
    // printf("add %d surfels.\n", add_surfel_num);
    // fuse_timer.end();

    // PoseElement this_pose_element;
    // this_pose_element.cam_pose = pose_input;
    // this_pose_element.cam_stamp = stamp;
    // poses_database.push_back(this_pose_element);
// }
